// matvect_mult.cu

#include <hip/hip_runtime.h>
#include <iostream>

// CUDA Kernel for matrix-vector multiplication
__global__ void matvecMulKernel(float* matrix, float* vector, float* result, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N) {
        float dot_product = 0.0f;
        for (int col = 0; col < N; col++) {
            dot_product += matrix[row * N + col] * vector[col];
        }
        result[row] = dot_product;
    }
}

// Host function to call the CUDA kernel
extern "C" void matvecMul(float* matrix, float* vector, float* result, int N) {
    float *d_matrix, *d_vector, *d_result;

    // Allocate memory on the GPU
    hipMalloc(&d_matrix, N * N * sizeof(float));
    hipMalloc(&d_vector, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));

    // Copy data from host (CPU) to device (GPU)
    hipMemcpy(d_matrix, matrix, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with N threads, 256 per block
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    matvecMulKernel<<<numBlocks, blockSize>>>(d_matrix, d_vector, d_result, N);

    // Copy the result back to the host (CPU)
    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);
}
